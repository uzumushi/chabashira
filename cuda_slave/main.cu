#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <string>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "sha256_revision.cuh"
#include <dirent.h>
#include <ctype.h>

#define THREAD_TOTAL 1052

dim3 grid_num(6);
dim3 block_num(192);
  

__device__ bool checkContinuousZeros(const char* str,int n_zeros){

    for (int i = 0; i < n_zeros; ++i){
        if(str[i] != '0')
            return false;
    }

    return true;
}

__device__ void addNonceToBlock(BLOCK_DATA* myblock,unsigned nonce){
	
	for(int i=7;i >= 0 ;i--){
		unsigned tmp = nonce >> (4*i);
		tmp &= 0x0f;
		myblock->data[myblock->size-8+(7-i)] = (10 <= tmp)*('a' + (tmp - 10)) + (10 > tmp)*('0' + tmp);

		
	}

}

__global__ void sha256_cuda(BLOCK_DATA * block,unsigned nonce_start,unsigned n_zero,bool * f_finish ,unsigned *d_answers) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	BYTE previous_hash[64];
	BYTE double_hash[64];
	BYTE digest[64];
	bool get_ans =false;
	unsigned  nonce= nonce_start + i;
	BLOCK_DATA myblock;

	memcpy(myblock.data,block->data,block->size);
	myblock.size = block->size + 8;	

	do{
		SHA256_CTX ctx;
		
		addNonceToBlock(&myblock,nonce);		

		sha256_init(&ctx);
		sha256_update(&ctx, myblock.data, myblock.size);
		sha256_final(&ctx, digest);
		
		hashStrCpy(previous_hash,digest);		

		sha256_init(&ctx);
		sha256_update(&ctx, previous_hash , 64);
		sha256_final(&ctx, digest);

		hashStrCpy(double_hash,digest);

		get_ans = checkContinuousZeros((char*)double_hash,n_zero);

	}while( get_ans == false && *f_finish == false && (nonce = nonce +THREAD_TOTAL));

	
	if(*f_finish ==false){
		*f_finish = true;
		d_answers[i] = nonce;
	}
}


void pre_sha256() {
	// compy symbols
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}



// * JOB_init(BYTE * data, long size) {
//	JOB * j;
//	checkCudaErrors(hipMallocManaged(&j, sizeof(JOB)));	//j = (JOB *)malloc(sizeof(JOB));
//	checkCudaErrors(hipMallocManaged(&(j->data), size));
//	j->data = data;
//	j->size = size;
//	for (int i = 0; i < 64; i++)
//	{
//		j->dige1152 (SM:6)st[i] = 0xff;
//	}
//	return j;
//}



int main() {
	
	int  i , n;
	BLOCK_DATA block;
	int nonce_start = 0;
	int n_zero = 3;
	unsigned answers[THREAD_TOTAL];	

	unsigned *d_answers;
	BLOCK_DATA *d_block;
	
	
	char data[] = "51528210305818912a0c5065e04921ae30a162641517c58dce4d4b4931e8853c5246820fa0d0000000896a97a80e4b869a93706ac86cc1cf8718f59fb5e4ffab78fc79c247e";

	strcpy((char*)(block.data),data);
	block.size = strlen(data);

	hipMalloc((void**)&d_block,sizeof(BLOCK_DATA) );
	hipMemcpy(d_block,&block,sizeof(BLOCK_DATA),hipMemcpyHostToDevice);
	
	hipMalloc((void**)&d_answers,sizeof(unsigned)*THREAD_TOTAL);
	hipMemset(d_answers,0,sizeof(unsigned)*THREAD_TOTAL); 	


	bool *f_finish;
	hipMalloc((void**)&f_finish,sizeof(bool));
	hipMemset(f_finish,false,1);
	sha256_cuda <<< grid_num,block_num  >>> (d_block,nonce_start,n_zero,f_finish,d_answers);

	hipDeviceSynchronize();

	hipMemcpy(answers,d_answers,sizeof(unsigned)*THREAD_TOTAL,hipMemcpyDeviceToHost);
		
	for(int i=0;i<THREAD_TOTAL;i++){
		printf("%x\n",answers[i]);
	}
	hipDeviceReset();
	return 0;
}
